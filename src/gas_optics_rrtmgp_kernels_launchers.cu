#include "hip/hip_runtime.h"
#include <chrono>
#include <functional>
#include <iostream>
#include <iomanip>

#include "gas_optics_rrtmgp_kernels_cuda.h"
#include "tools_gpu.h"
#include "tuner.h"


namespace
{
    #include "gas_optics_rrtmgp_kernels.cu"
}


namespace Gas_optics_rrtmgp_kernels_cuda
{
    void reorder123x321(
            const int ni, const int nj, const int nk,
            const Float* arr_in, Float* arr_out)
    {
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid(ni, nj, nk);
        dim3 block;

        if (tunings.count("reorder123x321_kernel") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "reorder123x321_kernel",
                dim3(ni, nj, nk),
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                reorder123x321_kernel,
                ni, nj, nk, arr_in, arr_out);

            tunings["reorder123x321_kernel"].first = grid;
            tunings["reorder123x321_kernel"].second = block;
        }
        else
        {
            grid = tunings["reorder123x321_kernel"].first;
            block = tunings["reorder123x321_kernel"].second;
        }

        reorder123x321_kernel<<<grid, block>>>(
                ni, nj, nk, arr_in, arr_out);
    }


    void reorder12x21(
            const int ni, const int nj,
            const Float* arr_in, Float* arr_out)
    {
        const int block_i = 32;
        const int block_j = 16;

        const int grid_i = ni/block_i + (ni%block_i > 0);
        const int grid_j = nj/block_j + (nj%block_j > 0);

        dim3 grid_gpu(grid_i, grid_j);
        dim3 block_gpu(block_i, block_j);

        reorder12x21_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, arr_in, arr_out);
    }


    void zero_array(const int ni, const int nj, const int nk, Float* arr)
    {
        const int block_i = 32;
        const int block_j = 16;
        const int block_k = 1;

        const int grid_i = ni/block_i + (ni%block_i > 0);
        const int grid_j = nj/block_j + (nj%block_j > 0);
        const int grid_k = nk/block_k + (nk%block_k > 0);

        dim3 grid_gpu(grid_i, grid_j, grid_k);
        dim3 block_gpu(block_i, block_j, block_k);

        zero_array_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, nk, arr);

    }


    void zero_array(const int ni, Float* arr)
    {
        zero_array(ni, 1, 1, arr);
    }


    void zero_array(const int ni, const int nj, Float* arr)
    {
        zero_array(ni, nj, 1, arr);
    }


    void interpolation(
            const int ncol, const int nlay,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int* flavor,
            const Float* press_ref_log,
            const Float* temp_ref,
            Float press_ref_log_delta,
            Float temp_ref_min,
            Float temp_ref_delta,
            Float press_ref_trop_log,
            const Float* vmr_ref,
            const Float* play,
            const Float* tlay,
            Float* col_gas,
            int* jtemp,
            Float* fmajor, Float* fminor,
            Float* col_mix,
            Bool* tropo,
            int* jeta,
            int* jpress)
    {
        const int block_col  = 4;
        const int block_lay  = 2;
        const int block_flav = 16;

        const int grid_col  = ncol /block_col  + (ncol%block_col   > 0);
        const int grid_lay  = nlay /block_lay  + (nlay%block_lay   > 0);
        const int grid_flav = nflav/block_flav + (nflav%block_flav > 0);

        dim3 grid_gpu(grid_col, grid_lay, grid_flav);
        dim3 block_gpu(block_col, block_lay, block_flav);

        Float tmin = std::numeric_limits<Float>::min();
        interpolation_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, ngas, nflav, neta, npres, ntemp, tmin,
                flavor, press_ref_log, temp_ref,
                press_ref_log_delta, temp_ref_min,
                temp_ref_delta, press_ref_trop_log,
                vmr_ref, play, tlay,
                col_gas, jtemp, fmajor,
                fminor, col_mix, tropo,
                jeta, jpress);
    }


    void combine_abs_and_rayleigh(
            const int ncol, const int nlay, const int ngpt,
            const Float* tau_abs, const Float* tau_rayleigh,
            Float* tau, Float* ssa, Float* g)
    {
        Tuner_map& tunings = Tuner::get_map();

        Float tmin = std::numeric_limits<Float>::min();

        dim3 grid(ncol, nlay, ngpt);
        dim3 block;

        if (tunings.count("combine_abs_and_rayleigh_kernel") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "combine_abs_and_rayleigh_kernel",
                dim3(ncol, nlay, ngpt),
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96}, {1, 2, 4}, {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                combine_abs_and_rayleigh_kernel,
                ncol, nlay, ngpt, tmin,
                tau_abs, tau_rayleigh,
                tau, ssa, g);

            tunings["combine_abs_and_rayleigh_kernel"].first = grid;
            tunings["combine_abs_and_rayleigh_kernel"].second = block;
        }
        else
        {
            grid = tunings["combine_abs_and_rayleigh_kernel"].first;
            block = tunings["combine_abs_and_rayleigh_kernel"].second;
        }

        combine_abs_and_rayleigh_kernel<<<grid, block>>>(
                ncol, nlay, ngpt, tmin,
                tau_abs, tau_rayleigh,
                tau, ssa, g);
    }


    void compute_tau_rayleigh(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int* gpoint_flavor,
            const int* band_lims_gpt,
            const Float* krayl,
            int idx_h2o, const Float* col_dry, const Float* col_gas,
            const Float* fminor, const int* jeta,
            const Bool* tropo, const int* jtemp,
            Float* tau_rayleigh)
    {
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid(ncol, nlay);
        dim3 block;

        if (tunings.count("compute_tau_rayleigh_kernel") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "compute_tau_rayleigh_kernel",
                dim3(ncol, nlay),
                {1, 2, 4, 16, 24, 32, 48, 64, 96, 128, 256, 512, 1024}, {1, 2, 4, 8, 16}, {1},
                compute_tau_rayleigh_kernel,
                ncol, nlay, nbnd, ngpt,
                ngas, nflav, neta, npres, ntemp,
                gpoint_flavor,
                band_lims_gpt,
                krayl,
                idx_h2o, col_dry, col_gas,
                fminor, jeta,
                tropo, jtemp,
                tau_rayleigh);

            tunings["compute_tau_rayleigh_kernel"].first = grid;
            tunings["compute_tau_rayleigh_kernel"].second = block;
        }
        else
        {
            grid = tunings["compute_tau_rayleigh_kernel"].first;
            block = tunings["compute_tau_rayleigh_kernel"].second;
        }

        compute_tau_rayleigh_kernel<<<grid, block>>>(
                ncol, nlay, nbnd, ngpt,
                ngas, nflav, neta, npres, ntemp,
                gpoint_flavor,
                band_lims_gpt,
                krayl,
                idx_h2o, col_dry, col_gas,
                fminor, jeta,
                tropo, jtemp,
                tau_rayleigh);
    }


    struct Gas_optical_depths_minor_kernel
    {
        template<unsigned int I, unsigned int J, unsigned int K, class... Args>
        static void launch(dim3 grid, dim3 block, Args... args)
        {
            gas_optical_depths_minor_kernel<I, J, K><<<grid, block>>>(args...);
        }
    };


    void compute_tau_absorption(
            const int ncol, const int nlay, const int nband, const int ngpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int nminorlower, const int nminorklower,
            const int nminorupper, const int nminorkupper,
            const int idx_h2o,
            const int* gpoint_flavor,
            const int* band_lims_gpt,
            const Float* kmajor,
            const Float* kminor_lower,
            const Float* kminor_upper,
            const int* minor_limits_gpt_lower,
            const int* minor_limits_gpt_upper,
            const Bool* minor_scales_with_density_lower,
            const Bool* minor_scales_with_density_upper,
            const Bool* scale_by_complement_lower,
            const Bool* scale_by_complement_upper,
            const int* idx_minor_lower,
            const int* idx_minor_upper,
            const int* idx_minor_scaling_lower,
            const int* idx_minor_scaling_upper,
            const int* kminor_start_lower,
            const int* kminor_start_upper,
            const Bool* tropo,
            const Float* col_mix, const Float* fmajor,
            const Float* fminor, const Float* play,
            const Float* tlay, const Float* col_gas,
            const int* jeta, const int* jtemp,
            const int* jpress,
            Float* tau)
    {
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid_gpu_maj(ngpt, nlay, ncol);
        dim3 block_gpu_maj;

        if (tunings.count("gas_optical_depths_major_kernel") == 0)
        {
            Float* tau_tmp = Tools_gpu::allocate_gpu<Float>(ngpt*nlay*ncol);

            std::tie(grid_gpu_maj, block_gpu_maj) = tune_kernel(
                    "gas_optical_depths_major_kernel",
                    dim3(ngpt, nlay, ncol),
                    {1, 2, 4, 8, 16, 24, 32, 48, 64}, {1, 2, 4}, {8, 16, 24, 32, 48, 64, 96, 128, 256},
                    gas_optical_depths_major_kernel,
                    ncol, nlay, nband, ngpt,
                    nflav, neta, npres, ntemp,
                    gpoint_flavor, band_lims_gpt,
                    kmajor, col_mix, fmajor, jeta,
                    tropo, jtemp, jpress,
                    tau_tmp);

            Tools_gpu::free_gpu<Float>(tau_tmp);

            tunings["gas_optical_depths_major_kernel"].first = grid_gpu_maj;
            tunings["gas_optical_depths_major_kernel"].second = block_gpu_maj;
        }
        else
        {
            grid_gpu_maj = tunings["gas_optical_depths_major_kernel"].first;
            block_gpu_maj = tunings["gas_optical_depths_major_kernel"].second;
        }

        gas_optical_depths_major_kernel<<<grid_gpu_maj, block_gpu_maj>>>(
                ncol, nlay, nband, ngpt,
                nflav, neta, npres, ntemp,
                gpoint_flavor, band_lims_gpt,
                kmajor, col_mix, fmajor, jeta,
                tropo, jtemp, jpress,
                tau);

        // Lower
        int idx_tropo = 1;

        dim3 grid_gpu_min_1(1, nlay, ncol);
        dim3 block_gpu_min_1;

        if (tunings.count("gas_optical_depths_minor_kernel_lower") == 0)
        {
            Float* tau_tmp = Tools_gpu::allocate_gpu<Float>(ngpt*nlay*ncol);
            std::tie(grid_gpu_min_1, block_gpu_min_1) =
                tune_kernel_compile_time<Gas_optical_depths_minor_kernel>(
                        "gas_optical_depths_minor_kernel_lower",
                        dim3(1, nlay, ncol),
                        std::integer_sequence<unsigned int, 1, 2, 4, 8, 16>{},
                        std::integer_sequence<unsigned int, 1, 2, 4>{},
                        std::integer_sequence<unsigned int, 1, 2, 4, 8, 16, 32, 48, 64, 96, 128>{},
                        ncol, nlay, ngpt,
                        ngas, nflav, ntemp, neta,
                        nminorlower,
                        nminorklower,
                        idx_h2o, idx_tropo,
                        gpoint_flavor,
                        kminor_lower,
                        minor_limits_gpt_lower,
                        minor_scales_with_density_lower,
                        scale_by_complement_lower,
                        idx_minor_lower,
                        idx_minor_scaling_lower,
                        kminor_start_lower,
                        play, tlay, col_gas,
                        fminor, jeta, jtemp,
                        tropo, tau_tmp, nullptr);
            Tools_gpu::free_gpu<Float>(tau_tmp);

            tunings["gas_optical_depths_minor_kernel_lower"].first = grid_gpu_min_1;
            tunings["gas_optical_depths_minor_kernel_lower"].second = block_gpu_min_1;
        }
        else
        {
            grid_gpu_min_1 = tunings["gas_optical_depths_minor_kernel_lower"].first;
            block_gpu_min_1 = tunings["gas_optical_depths_minor_kernel_lower"].second;
        }

        run_kernel_compile_time<Gas_optical_depths_minor_kernel>(
                std::integer_sequence<unsigned int, 1, 2, 4, 8, 16>{},
                std::integer_sequence<unsigned int, 1, 2, 4>{},
                std::integer_sequence<unsigned int, 1, 2, 4, 8, 16, 32, 48, 64, 96, 128>{},
                grid_gpu_min_1, block_gpu_min_1,
                ncol, nlay, ngpt,
                ngas, nflav, ntemp, neta,
                nminorlower,
                nminorklower,
                idx_h2o, idx_tropo,
                gpoint_flavor,
                kminor_lower,
                minor_limits_gpt_lower,
                minor_scales_with_density_lower,
                scale_by_complement_lower,
                idx_minor_lower,
                idx_minor_scaling_lower,
                kminor_start_lower,
                play, tlay, col_gas,
                fminor, jeta, jtemp,
                tropo, tau, nullptr);


        // Upper
        idx_tropo = 0;

        dim3 grid_gpu_min_2(ngpt, nlay, ncol);
        dim3 block_gpu_min_2;

        if (tunings.count("gas_optical_depths_minor_kernel_upper") == 0)
        {
            Float* tau_tmp = Tools_gpu::allocate_gpu<Float>(ngpt*nlay*ncol);
            std::tie(grid_gpu_min_2, block_gpu_min_2) =
                tune_kernel_compile_time<Gas_optical_depths_minor_kernel>(
                        "gas_optical_depths_minor_kernel_upper",
                        dim3(1, nlay, ncol),
                        std::integer_sequence<unsigned int, 1, 2, 4, 8, 16>{},
                        std::integer_sequence<unsigned int, 1, 2, 4>{},
                        std::integer_sequence<unsigned int, 1, 2, 4, 8, 16, 32, 48, 64, 96, 128>{},
                        ncol, nlay, ngpt,
                        ngas, nflav, ntemp, neta,
                        nminorupper,
                        nminorkupper,
                        idx_h2o, idx_tropo,
                        gpoint_flavor,
                        kminor_upper,
                        minor_limits_gpt_upper,
                        minor_scales_with_density_upper,
                        scale_by_complement_upper,
                        idx_minor_upper,
                        idx_minor_scaling_upper,
                        kminor_start_upper,
                        play, tlay, col_gas,
                        fminor, jeta, jtemp,
                        tropo, tau_tmp, nullptr);
            Tools_gpu::free_gpu<Float>(tau_tmp);

            tunings["gas_optical_depths_minor_kernel_upper"].first = grid_gpu_min_2;
            tunings["gas_optical_depths_minor_kernel_upper"].second = block_gpu_min_2;
        }
        else
        {
            grid_gpu_min_2 = tunings["gas_optical_depths_minor_kernel_upper"].first;
            block_gpu_min_2 = tunings["gas_optical_depths_minor_kernel_upper"].second;
        }

        run_kernel_compile_time<Gas_optical_depths_minor_kernel>(
                std::integer_sequence<unsigned int, 1, 2, 4, 8, 16>{},
                std::integer_sequence<unsigned int, 1, 2, 4>{},
                std::integer_sequence<unsigned int, 1, 2, 4, 8, 16, 32, 48, 64, 96, 128>{},
                grid_gpu_min_2, block_gpu_min_2,
                ncol, nlay, ngpt,
                ngas, nflav, ntemp, neta,
                nminorupper,
                nminorkupper,
                idx_h2o, idx_tropo,
                gpoint_flavor,
                kminor_upper,
                minor_limits_gpt_upper,
                minor_scales_with_density_upper,
                scale_by_complement_upper,
                idx_minor_upper,
                idx_minor_scaling_upper,
                kminor_start_upper,
                play, tlay, col_gas,
                fminor, jeta, jtemp,
                tropo, tau, nullptr);
    }


    void compute_planck_source(
            const int ncol, const int nlay, const int nbnd, const int ngpt,
            const int nflav, const int neta, const int npres, const int ntemp,
            const int nPlanckTemp,
            const Float* tlay,
            const Float* tlev,
            const Float* tsfc,
            const int sfc_lay,
            const Float* fmajor,
            const int* jeta,
            const Bool* tropo,
            const int* jtemp,
            const int* jpress,
            const int* gpoint_bands,
            const int* band_lims_gpt,
            const Float* pfracin,
            const Float temp_ref_min, const Float totplnk_delta,
            const Float* totplnk,
            const int* gpoint_flavor,
            Float* sfc_src,
            Float* lay_src,
            Float* lev_src_inc,
            Float* lev_src_dec,
            Float* sfc_src_jac)
    {
        Tuner_map& tunings = Tuner::get_map();

        const Float delta_Tsurf = Float(1.);

        const int block_col = 32;
        const int block_lay = 4;

        const int grid_col = ncol/block_col + (ncol%block_col > 0);
        const int grid_lay = nlay/block_lay + (nlay%block_lay > 0);

        dim3 grid_gpu(grid_col, grid_lay);
        dim3 block_gpu(block_col, block_lay);
        
        if (tunings.count("Planck_source_kernel") == 0)
        {
            std::tie(grid_gpu, block_gpu) = tune_kernel(
                    "Planck_source_kernel",
                    dim3(ncol, nlay),
                    {1, 2, 4, 8, 16, 32, 48, 64, 96, 128, 256, 512},
                    {1, 2, 4, 8, 16, 32, 48, 64, 96, 128, 256, 512},
                    {1},
                    Planck_source_kernel,
                    ncol, nlay, nbnd, ngpt,
                    nflav, neta, npres, ntemp, nPlanckTemp,
                    tlay, tlev, tsfc, sfc_lay,
                    fmajor, jeta, tropo, jtemp,
                    jpress, gpoint_bands, band_lims_gpt,
                    pfracin, temp_ref_min, totplnk_delta,
                    totplnk, gpoint_flavor,
                    delta_Tsurf, sfc_src, lay_src,
                    lev_src_inc, lev_src_dec,
                    sfc_src_jac);
            
            tunings["Planck_source_kernel"].first = grid_gpu;
            tunings["Planck_source_kernel"].second = block_gpu;
        }
        else
        {
            grid_gpu = tunings["Planck_source_kernel"].first;
            block_gpu = tunings["Planck_source_kernel"].second;
        }

        Planck_source_kernel<<<grid_gpu, block_gpu>>>(
                ncol, nlay, nbnd, ngpt,
                nflav, neta, npres, ntemp, nPlanckTemp,
                tlay, tlev, tsfc, sfc_lay,
                fmajor, jeta, tropo, jtemp,
                jpress, gpoint_bands, band_lims_gpt,
                pfracin, temp_ref_min, totplnk_delta,
                totplnk, gpoint_flavor,
                delta_Tsurf,
                sfc_src, lay_src,
                lev_src_inc, lev_src_dec,
                sfc_src_jac);
    }
}
