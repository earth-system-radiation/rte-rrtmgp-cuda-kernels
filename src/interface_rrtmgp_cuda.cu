#include <openacc.h>
#include <cstdio>
#include <stdexcept>

#include "types.h"
#include "tools_gpu.h"
#include "gas_optics_rrtmgp_kernels_cuda.h"


namespace
{
    template<typename T> T* acc_to_cuda(T* ptr) { return static_cast<T*>(acc_deviceptr(ptr)); }
}


extern "C"
{
    void rrtmgp_interpolation(
            int* ncol, int* nlay,
            int* ngas, int* nflav, int* neta, int* npres, int* ntemp,
            int* flavor,
            Float* press_ref_log,
            Float* temp_ref,
            Float* press_ref_log_delta,
            Float* temp_ref_min,
            Float* temp_ref_delta,
            Float* press_ref_trop_log,
            Float* vmr_ref,
            Float* play,
            Float* tlay,
            Float* col_gas,
            int* jtemp,
            Float* fmajor, Float* fminor,
            Float* col_mix,
            Bool* tropo,
            int* jeta,
            int* jpress)
    {
        // printf("CvH: interpolation CUDA\n");
        Gas_optics_rrtmgp_kernels_cuda::interpolation(
                *ncol, *nlay,
                *ngas, *nflav, *neta, *npres, *ntemp,
                acc_to_cuda(flavor),
                acc_to_cuda(press_ref_log),
                acc_to_cuda(temp_ref),
                *press_ref_log_delta,
                *temp_ref_min,
                *temp_ref_delta,
                *press_ref_trop_log,
                acc_to_cuda(vmr_ref),
                acc_to_cuda(play),
                acc_to_cuda(tlay),
                acc_to_cuda(col_gas),
                acc_to_cuda(jtemp),
                acc_to_cuda(fmajor), acc_to_cuda(fminor),
                acc_to_cuda(col_mix),
                acc_to_cuda(tropo),
                acc_to_cuda(jeta),
                acc_to_cuda(jpress));

        cuda_safe_call(hipStreamSynchronize(0));
    }


    void rrtmgp_compute_tau_absorption(
            int* ncol, int* nlay, int* nband, int* ngpt,
            int* ngas, int* nflav, int* neta, int* npres, int* ntemp,
            int* nminorlower, int* nminorklower,
            int* nminorupper, int* nminorkupper,
            int* idx_h2o,
            int* gpoint_flavor,
            int* band_lims_gpt,
            Float* kmajor,
            Float* kminor_lower,
            Float* kminor_upper,
            int* minor_limits_gpt_lower,
            int* minor_limits_gpt_upper,
            Bool* minor_scales_with_density_lower,
            Bool* minor_scales_with_density_upper,
            Bool* scale_by_complement_lower,
            Bool* scale_by_complement_upper,
            int* idx_minor_lower,
            int* idx_minor_upper,
            int* idx_minor_scaling_lower,
            int* idx_minor_scaling_upper,
            int* kminor_start_lower,
            int* kminor_start_upper,
            Bool* tropo,
            Float* col_mix, Float* fmajor,
            Float* fminor, Float* play,
            Float* tlay, Float* col_gas,
            int* jeta, int* jtemp,
            int* jpress, Float* tau)
    {
        // printf("CvH: compute_tau_absorption CUDA\n");
        Gas_optics_rrtmgp_kernels_cuda::compute_tau_absorption(
                *ncol, *nlay, *nband, *ngpt,
                *ngas, *nflav, *neta, *npres, *ntemp,
                *nminorlower, *nminorklower,
                *nminorupper, *nminorkupper,
                *idx_h2o,
                acc_to_cuda(gpoint_flavor),
                acc_to_cuda(band_lims_gpt),
                acc_to_cuda(kmajor),
                acc_to_cuda(kminor_lower),
                acc_to_cuda(kminor_upper),
                acc_to_cuda(minor_limits_gpt_lower),
                acc_to_cuda(minor_limits_gpt_upper),
                acc_to_cuda(minor_scales_with_density_lower),
                acc_to_cuda(minor_scales_with_density_upper),
                acc_to_cuda(scale_by_complement_lower),
                acc_to_cuda(scale_by_complement_upper),
                acc_to_cuda(idx_minor_lower),
                acc_to_cuda(idx_minor_upper),
                acc_to_cuda(idx_minor_scaling_lower),
                acc_to_cuda(idx_minor_scaling_upper),
                acc_to_cuda(kminor_start_lower),
                acc_to_cuda(kminor_start_upper),
                acc_to_cuda(tropo),
                acc_to_cuda(col_mix), acc_to_cuda(fmajor),
                acc_to_cuda(fminor), acc_to_cuda(play),
                acc_to_cuda(tlay), acc_to_cuda(col_gas),
                acc_to_cuda(jeta), acc_to_cuda(jtemp),
                acc_to_cuda(jpress), acc_to_cuda(tau));

        cuda_safe_call(hipStreamSynchronize(0));
    }


    void rrtmgp_compute_tau_rayleigh(
            int* ncol, int* nlay, int* nband, int* ngpt,
            int* ngas, int* nflav, int* neta, int* npres, int* ntemp,
            int* gpoint_flavor,
            int* band_lims_gpt,
            Float* krayl,
            int* idx_h2o, Float* col_dry, Float* col_gas,
            Float* fminor, int* jeta,
            Bool* tropo, int* jtemp,
            Float* tau_rayleigh)
    {
        // printf("CvH: compute_tau_rayleigh CUDA\n");
        Gas_optics_rrtmgp_kernels_cuda::compute_tau_rayleigh(
                *ncol, *nlay, *nband, *ngpt,
                *ngas, *nflav, *neta, *npres, *ntemp,
                acc_to_cuda(gpoint_flavor),
                acc_to_cuda(band_lims_gpt),
                acc_to_cuda(krayl),
                *idx_h2o, acc_to_cuda(col_dry), acc_to_cuda(col_gas),
                acc_to_cuda(fminor), acc_to_cuda(jeta),
                acc_to_cuda(tropo), acc_to_cuda(jtemp),
                acc_to_cuda(tau_rayleigh));

        cuda_safe_call(hipStreamSynchronize(0));
    }


    void rrtmgp_compute_Planck_source(
            int* ncol, int* nlay, int* nbnd, int* ngpt,
            int* nflav, int* neta, int* npres, int* ntemp,
            int* nPlanckTemp,
            Float* tlay,
            Float* tlev,
            Float* tsfc,
            int* sfc_lay,
            Float* fmajor,
            int* jeta,
            Bool* tropo,
            int* jtemp,
            int* jpress,
            int* gpoint_bands,
            int* band_lims_gpt,
            Float* pfracin,
            Float* temp_ref_min, Float* totplnk_delta,
            Float* totplnk,
            int* gpoint_flavor,
            Float* sfc_src,
            Float* lay_src,
            Float* lev_src_inc,
            Float* lev_src_dec,
            Float* sfc_src_jac)
    {
        // printf("CvH: compute_planck_source CUDA\n");
        Gas_optics_rrtmgp_kernels_cuda::compute_planck_source(
                *ncol, *nlay, *nbnd, *ngpt,
                *nflav, *neta, *npres, *ntemp,
                *nPlanckTemp,
                acc_to_cuda(tlay),
                acc_to_cuda(tlev),
                acc_to_cuda(tsfc),
                *sfc_lay,
                acc_to_cuda(fmajor),
                acc_to_cuda(jeta),
                acc_to_cuda(tropo),
                acc_to_cuda(jtemp),
                acc_to_cuda(jpress),
                acc_to_cuda(gpoint_bands),
                acc_to_cuda(band_lims_gpt),
                acc_to_cuda(pfracin),
                *temp_ref_min, *totplnk_delta,
                acc_to_cuda(totplnk),
                acc_to_cuda(gpoint_flavor),
                acc_to_cuda(sfc_src),
                acc_to_cuda(lay_src),
                acc_to_cuda(lev_src_inc),
                acc_to_cuda(lev_src_dec),
                acc_to_cuda(sfc_src_jac));

        cuda_safe_call(hipStreamSynchronize(0));
    }

    void zero_array_1D(int* ni, Float* array)
    {
        Gas_optics_rrtmgp_kernels_cuda::zero_array(*ni, acc_to_cuda(array));
    }

    void zero_array_2D(int* ni, int* nj, Float* array)
    {
        Gas_optics_rrtmgp_kernels_cuda::zero_array(*ni, *nj, acc_to_cuda(array));
    }

    void zero_array_3D(int* ni, int* nj, int* nk, Float* array)
    {
        Gas_optics_rrtmgp_kernels_cuda::zero_array(*ni, *nj, *nk, acc_to_cuda(array));
    }

    void zero_array_4D(int* ni, int* nj, int* nk, int* nl, Float* array)
    {
        throw std::runtime_error("zero_array_4D is not implemented in CUDA");
    }
}
