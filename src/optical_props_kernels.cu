#include "hip/hip_runtime.h"
/*
 * This file is part of a C++ interface to the Radiative Transfer for Energetics (RTE)
 * and Rapid Radiative Transfer Model for GCM applications Parallel (RRTMGP).
 *
 * The original code is found at https://github.com/earth-system-radiation/rte-rrtmgp.
 *
 * Contacts: Robert Pincus and Eli Mlawer
 * email: rrtmgp@aer.com
 *
 * Copyright 2015-2020,  Atmospheric and Environmental Research and
 * Regents of the University of Colorado.  All right reserved.
 *
 * This C++ interface can be downloaded from https://github.com/earth-system-radiation/rte-rrtmgp-cpp
 *
 * Contact: Chiel van Heerwaarden
 * email: chiel.vanheerwaarden@wur.nl
 *
 * Copyright 2020, Wageningen University & Research.
 *
 * Use and duplication is permitted under the terms of the
 * BSD 3-clause license, see http://opensource.org/licenses/BSD-3-Clause
 *
 */

#ifndef kernel_tuner
const int loop_unroll_factor_nbnd = 2;
#endif


__global__
void increment_1scalar_by_1scalar_kernel(
            const int ncol, const int nlay, const int ngpt,
            Float* __restrict__ tau1, const Float* __restrict__ tau2)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const int idx = icol + ilay*ncol + igpt*ncol*nlay;
        tau1[idx] = tau1[idx]+tau2[idx];
    }
}


__global__
void increment_2stream_by_2stream_kernel(
            const int ncol, const int nlay, const int ngpt, const Float eps,
            Float* __restrict__ tau1, Float* __restrict__ ssa1, Float* __restrict__ g1,
            const Float* __restrict__ tau2, const Float* __restrict__ ssa2, const Float* __restrict__ g2)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const int idx = icol + ilay*ncol + igpt*ncol*nlay;
        const Float tau1_value = tau1[idx];
        const Float tau2_value = tau2[idx];
        const Float tau12 = tau1_value + tau2_value;
        const Float ssa1_value = ssa1[idx];
        const Float ssa2_value = ssa2[idx];
        const Float tauscat12 = (tau1_value * ssa1_value) + (tau2_value * ssa2_value);

        g1[idx] = ((tau1_value * ssa1_value * g1[idx]) + (tau2_value * ssa2[idx] * g2[idx])) / max(tauscat12, eps);
        ssa1[idx] = tauscat12 / max(eps, tau12);
        tau1[idx] = tau12;
    }
}


__global__
void inc_1scalar_by_1scalar_bybnd_kernel(
            const int ncol, const int nlay, const int ngpt,
            Float* __restrict__ tau1, const Float* __restrict__ tau2,
            const int nbnd, const int* __restrict__ band_lims_gpt)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const int idx_gpt = icol + ilay*ncol + igpt*nlay*ncol;

        #pragma unroll loop_unroll_factor_nbnd
        for (int ibnd=0; ibnd<nbnd; ++ibnd)
        {
            if ( ((igpt+1) >= band_lims_gpt[ibnd*2]) && ((igpt+1) <= band_lims_gpt[ibnd*2+1]) )
            {
                const int idx_bnd = icol + ilay*ncol + ibnd*nlay*ncol;

                tau1[idx_gpt] = tau1[idx_gpt] + tau2[idx_bnd];
            }
        }
    }
}


__global__
void inc_2stream_by_2stream_bybnd_kernel(
            const int ncol, const int nlay, const int ngpt, const Float eps,
            Float* __restrict__ tau1, Float* __restrict__ ssa1, Float* __restrict__ g1,
            const Float* __restrict__ tau2, const Float* __restrict__ ssa2, const Float* __restrict__ g2,
            const int nbnd, const int* __restrict__ band_lims_gpt)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const int idx_gpt = icol + ilay*ncol + igpt*nlay*ncol;

        #pragma unroll loop_unroll_factor_nbnd
        for (int ibnd=0; ibnd<nbnd; ++ibnd)
        {
            if ( ((igpt+1) >= band_lims_gpt[ibnd*2]) && ((igpt+1) <= band_lims_gpt[ibnd*2+1]) )
            {
                const int idx_bnd = icol + ilay*ncol + ibnd*nlay*ncol;
                const Float tau1_value = tau1[idx_gpt];
                const Float tau2_value = tau2[idx_bnd];
                const Float ssa1_value = ssa1[idx_gpt];
                const Float ssa2_value = ssa2[idx_bnd];
                const Float tau12 = tau1_value + tau2_value;
                const Float tauscat12 = (tau1_value * ssa1_value) + (tau2_value * ssa2_value);

                g1[idx_gpt] = ((tau1_value * ssa1_value * g1[idx_gpt]) + (tau2_value * ssa2_value * g2[idx_bnd])) / max(tauscat12, eps);
                ssa1[idx_gpt] = tauscat12 / max(eps, tau12);
                tau1[idx_gpt] = tau12;
            }
        }
    }
}


__global__
void delta_scale_2str_k_kernel(
            const int ncol, const int nlay, const int ngpt, const Float eps,
            Float* __restrict__ tau, Float* __restrict__ ssa, Float* __restrict__ g)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const int idx = icol + ilay*ncol + igpt*nlay*ncol;
        const Float g_value = g[idx];
        const Float ssa_value = ssa[idx];
        const Float f = g_value * g_value;
        const Float wf = ssa_value * f;

        tau[idx] *= (Float(1.) - wf);
        ssa[idx] = (ssa_value - wf) / max(eps,(Float(1.)-wf));
        g[idx] = (g_value - f) / max(eps,(Float(1.)-f));

    }
}
